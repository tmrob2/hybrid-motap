#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include "thrust/device_vector.h"
#include "thrust/device_ptr.h"
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

//int MAX_ITERATIONS = 1000;
//const int MAX_UNSTABLE = 30;
/*
#######################################################################
#                           KERNELS                                   #
#######################################################################
*/

__global__ void max_value(
    float *y,
    int *enabled_actions,
    int *adj_sidx,
    float *xnew,
    float *xold,
    int *pi,
    float *stable,
    float epsilon,
    int N
    ) {
    // The purpose of this kernel is to do effective row-wise comparison of values
    // to determine the new policy and the new value vector without copy of 
    // data from the GPU to CPU
    //
    // It is recognised that this code will be slow due to memory segmentation
    // and cache access, but this should in theory be faster then sending data
    // back and forth between the GPU and CPU
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        float max_value = -INFINITY;
        int argmax_a = -1;
        for (int k = 0; k < enabled_actions[tid]; k++) {
            if (y[adj_sidx[tid] + k] > max_value) {
                max_value = y[adj_sidx[tid] + k];
                argmax_a = k;
            }
        }
        xnew[tid] = max_value;
        stable[tid] = 0.;
        if (max_value - xold[tid] > epsilon) {
            stable[tid] = 1.0;
            pi[tid] = argmax_a;
        }
    }
}

__global__ void abs_diff(float *a, float *b, float *c, int *unstable, int m, int max_unstable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // HANDLE THE DATA AT THIS INDEX
    if (tid < m) {
        // compute the absolute diff between two elems
        if (fabsf(b[tid] - a[tid]) < c[tid] || a[tid] == 0.) {
            unstable[tid] = 0;
        } else {
            unstable[tid]++;
        }
        c[tid] = fabsf(b[tid] - a[tid]);
        if (unstable[tid] > max_unstable && a[tid] < 0) {
            a[tid] = -INFINITY;
        }
    } 
}

__global__ void mobj_abs_diff(float *x, float *y, float *eps_capture, int *unstable, 
    int obj, int N, int max_unstable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        if (fabsf(x[tid] - y[tid]) < eps_capture[obj * N + tid] || y[tid] == 0.) {
            unstable[obj * N + tid] = 0;
        } else {
            unstable[obj * N + tid]++;
        }
        eps_capture[obj * N + tid] = fabs(x[tid] - y[tid]);
        if (unstable[obj * N + tid] > max_unstable && y[tid] < 0) {
            y[tid] = -INFINITY;
        }
    }
}

__global__ void change_elem(float *arr, int idx, int val) {
    arr[idx] = val;
}

__global__ void copy_elems(float *dest, int begin_idx, float *src, int begin_cp, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        dest[tid + begin_idx] = src[tid + begin_cp];
    }
}

void copy_elems_launcher(float *dest, int begin_idx, float *src, int begin_cp, int N) {
    int blockSize = 0;    // The launch configurator returned block size
    int minGridSize;  // The maximum grid size needed to achieve max
                      // maximum occupancy
    int gridSize;     // The grid size needed, based on the input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, max_value, 0, 0);

    // Round up according to array size
    gridSize = (N + blockSize - 1) / blockSize;

    copy_elems<<<gridSize, blockSize>>>(dest, begin_idx, src, begin_cp, N);
}

void max_value_launcher(float *y, int*enabled_actions, int *adj_sidx, float *xnew,
    float *xold, int *pi, float *stable, float epsilon, int N
) {
    int blockSize = 0;    // The launch configurator returned block size
    int minGridSize;  // The maximum grid size needed to achieve max
                      // maximum occupancy
    int gridSize;     // The grid size needed, based on the input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, max_value, 0, 0);

    // Round up according to array size
    gridSize = (N + blockSize - 1) / blockSize;

    max_value<<<gridSize, blockSize>>>(y, enabled_actions, adj_sidx, xnew, xold, pi,
        stable, epsilon, N
    );
}

void mobj_abs_diff_launcher(float *x, float*y, float *eps_capture, int *unstable, 
    int obj, int N, int max_unstable) {
    int blockSize = 0;    // The launch configurator returned block size
    int minGridSize = 0;  // The maximum grid size needed to achieve max
                      // maximum occupancy
    int gridSize;     // The grid size needed, based on the input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, max_value, 0, 0);

    // Round up according to array size
    gridSize = (N + blockSize - 1) / blockSize;

    mobj_abs_diff<<<gridSize, blockSize>>>(x, y, eps_capture, unstable, obj, N, max_unstable);
}

void abs_diff_launcher(float *a, float *b, float* c, int *unstable, int m, int max_unstable) {
    int blockSize = 0;    // The launch configurator returned block size
    int minGridSize;  // The maximum grid size needed to achieve max
                      // maximum occupancy
    int gridSize;     // The grid size needed, based on the input size

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, abs_diff, 0, 0);

    // Round up according to array size
    gridSize = (m + blockSize - 1) / blockSize;

    abs_diff<<<gridSize, blockSize>>>(a, b, c, unstable, m, max_unstable);
}

/*
#######################################################################
#                              CUDA                                   #
#######################################################################
*/

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

extern "C" {

int warm_up_gpu() {
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    hipblasHandle_t blashandle;
    CHECK_CUBLAS(hipblasCreate(&blashandle));

    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUBLAS(hipblasDestroy(blashandle));
    return 0.;
}

int initial_policy_value(
    int pm,
    int pn,
    int pnz,
    int * pi,
    int * pj,
    float * px,
    int pi_size,
    int rm,
    int rn,
    int rnz,
    int *ri,
    int *rj,
    float *rx,
    int ri_size,
    float *x,
    float *y,
    float *w,
    float *rmv,
    int *unstable,
    float eps,
    int max_iter,
    int max_unstable
    ) {
    /* 
    Get the COO matrix into sparsescoo fmt

    Then multiply the COO by the initial value vector

    The rewards matrix is also sparse so it will need a sparse matrix descr
    as well. Multiply R by a repeated weight vector in the number 
    of prods and actions

    Finally sum the result

    This should happen in a loop until convergence

    I also want to do some wall timing to see some statistics on 
    the GPU 
    */
    // build the sparse transition matrix first

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    hipblasHandle_t blashandle;
    CHECK_CUBLAS(hipblasCreate(&blashandle));


    hipsparseSpMatDescr_t descrP = NULL;
    hipsparseSpMatDescr_t descrR = NULL;

    // allocated the device memory for the COO matrix

    // ----------------------------------------------------------------
    //                       Transition Matrix
    // ----------------------------------------------------------------

    //allocate dCsrRowPtr, dCsrColPtr, dCsrValPtr
    int *dPCsrRowPtr, *dPCsrColPtr;
    float *dPCsrValPtr;

    // allocate device memory to store the sparse CSR 
    hipMalloc((void **)&dPCsrValPtr, sizeof(float) * pnz);
    hipMalloc((void **)&dPCsrRowPtr, sizeof(int) * pi_size);
    hipMalloc((void **)&dPCsrColPtr, sizeof(int) * pnz);

    hipMemcpy(dPCsrValPtr, px, sizeof(float) * pnz, hipMemcpyHostToDevice);
    hipMemcpy(dPCsrColPtr, pj, sizeof(int) * pnz, hipMemcpyHostToDevice);
    hipMemcpy(dPCsrRowPtr, pi, sizeof(int) * pi_size, hipMemcpyHostToDevice);
    
    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrP, // MATRIX DESCRIPTION
        pm, // NUMBER OF ROWS
        pn, // NUMBER OF COLS
        pnz, // NUMBER OF NON ZERO VALUES
        dPCsrRowPtr, // ROWS OFFSETS
        dPCsrColPtr, // COL INDICES
        dPCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));
    
    // ----------------------------------------------------------------
    //                       Rewards Matrix
    // ----------------------------------------------------------------
    
    int *dRCsrRowPtr, *dRCsrColPtr;
    float *dRCsrValPtr;

    // allocate device memory to store the sparse CSR 
    hipMalloc((void **)&dRCsrValPtr, sizeof(float) * rnz);
    hipMalloc((void **)&dRCsrRowPtr, sizeof(int) * ri_size);
    hipMalloc((void **)&dRCsrColPtr, sizeof(int) * rnz);
    hipMemcpy(dRCsrValPtr, rx, sizeof(float) * rnz, hipMemcpyHostToDevice);
    hipMemcpy(dRCsrColPtr, rj, sizeof(int) * rnz, hipMemcpyHostToDevice);
    hipMemcpy(dRCsrRowPtr, ri, sizeof(int) * ri_size, hipMemcpyHostToDevice);

    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrR, // MATRIX DESCRIPTION
        rm, // NUMBER OF ROWS
        rn, // NUMBER OF COLS
        rnz, // NUMBER OF NON ZERO VALUES
        dRCsrRowPtr, // ROWS OFFSETS
        dRCsrColPtr, // COL INDICES
        dRCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));

    // ----------------------------------------------------------------
    //                      Start of VI
    // ----------------------------------------------------------------

    // --------------TRANSITION MATRIX MULTIPLICATION SETUP------------
    
    float alpha = 1.0;
    float beta = 1.0;
    float *epsilon = (float*) malloc(pm * sizeof(float));
    //int iepsilon;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecX, vecY;
    float *dX, *dY, *dZ, *dStaticY, *dOutput;
    int *dUnstable;
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    // Allocate the device memory
    hipMalloc((void**)&dX, pm * sizeof(float));
    hipMalloc((void**)&dOutput, pm * sizeof(float));
    hipMalloc((void**)&dY, pm * sizeof(float));
    hipMalloc((void**)&dZ, pm * sizeof(float));
    hipMalloc((void**)&dStaticY, pm * sizeof(float));
    hipMalloc((void**)&dUnstable, pm * sizeof(float));
    //hipMalloc((void**)&d_eps, sizeof(float));

    // create a initial Y vector
    float *static_y = (float*) calloc(pm, sizeof(float));
    
    // copy the vector from host memory to device memory
    hipMemcpy(dX, x, pn * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dStaticY, static_y, pm * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dUnstable, unstable, pm * sizeof(float), hipMemcpyHostToDevice);

    // create a dense vector on device memory
    hipsparseCreateDnVec(&vecX, pn, dX, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, pm, dY, HIP_R_32F);

    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descrP, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    
    // --------------REWARDS MATRIX MULTIPLICATION SETUP---------------

    float alphaR = 1.0;
    float betaR = 1.0;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecW, vecRMv;
    float *dRw, *dRMv;
    void* dBufferR = NULL;
    size_t bufferSizeR = 0;

    //float *rmv = (float*) calloc(rm, sizeof(float));

    hipMalloc((void**)&dRw, rn * sizeof(float));
    hipMalloc((void**)&dRMv, rm * sizeof(float));

    // copy the vector from host memory to device memory
    hipMemcpy(dRw, w, rn * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dRMv, rmv, rm * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(dRstaticMx, rmv, rm * sizeof(float), hipMemcpyHostToDevice);

    // create a dense vector on device memory
    hipsparseCreateDnVec(&vecW, rn, dRw, HIP_R_32F);
    hipsparseCreateDnVec(&vecRMv, rm, dRMv, HIP_R_32F);

    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeR);
    hipMalloc(&dBufferR, bufferSizeR);

    // ALGORITHM LOOP

    // Copy the zero vector to initialise Y -> captures A.x result 
    // for transition matrix
    //csparseDnVecSetValues(vecY, dY);
    //hipblasScopy(blashandle, pm, dYStatic, 1, dY, 1);
    // copy the static Y vector to initialise Y
    CHECK_CUSPARSE(hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F, 
        HIPSPARSE_MV_ALG_DEFAULT, dBufferR));

    float maxeps;
    maxeps = 0.0f;

    for (int algo_i = 0; algo_i < max_iter; algo_i ++) {

        CHECK_CUSPARSE(hipsparseSpMV(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, descrP, vecX, &beta, vecY, HIP_R_32F, 
            HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

        // push this into the algorithm loop


        // ---------------------SUM DENSE VECTORS-------------------------

        /* 
        The gpu memory should already be allocated, i.e. we are summing
        dY + dRMv
        */
        CHECK_CUBLAS(hipblasSaxpy(blashandle, pm, &alpha, dRMv, 1, dY, 1));
        
        // ---------------------COMPUTE EPSILON---------------------------

        // what is the difference between dY and dX

        // EPSILON COMPUTATION
        abs_diff_launcher(dY, dX, dZ, dUnstable, pm, max_unstable);
        //CHECK_CUBLAS(hipblasIsamax(blashandle, pm, dZ, 1, &iepsilon));

        thrust::device_ptr<float> dev_ptr(dZ);
        maxeps = *thrust::max_element(thrust::device, dev_ptr, dev_ptr + pm);

        CHECK_CUBLAS(hipblasScopy(blashandle, pm, dY, 1, dX, 1));
        // RESET Y
        CHECK_CUBLAS(hipblasScopy(blashandle, pm, dStaticY, 1, dY, 1));
        //std::cout << "EPS_TEST " << "THRUST "<< maxeps << std::endl;
        if (maxeps < eps || isnan(maxeps)) {
            //printf("INITIAL POLICY GENERATED; EPS TOL REACHED in %i ITERATIONS\n", algo_i);
            break;
        }
    }
    
    CHECK_CUDA(hipMemcpy(y, dX, pm *sizeof(float), hipMemcpyDeviceToHost));
    
    //hipMemcpy(rmv, dRMv, rm *sizeof(float), hipMemcpyDeviceToHost);
    //destroy the vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(descrP));
    CHECK_CUSPARSE(hipsparseDestroySpMat(descrR));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRMv));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecW));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUBLAS(hipblasDestroy(blashandle));

    // Free the device memory
    CHECK_CUDA(hipFree(dPCsrColPtr));
    CHECK_CUDA(hipFree(dPCsrRowPtr));
    CHECK_CUDA(hipFree(dPCsrValPtr));
    CHECK_CUDA(hipFree(dRCsrColPtr));
    CHECK_CUDA(hipFree(dRCsrRowPtr));
    CHECK_CUDA(hipFree(dRCsrValPtr));
    //hipFree(d_eps);
    CHECK_CUDA(hipFree(dX));
    CHECK_CUDA(hipFree(dY));
    CHECK_CUDA(hipFree(dStaticY));
    CHECK_CUDA(hipFree(dUnstable));
    CHECK_CUDA(hipFree(dZ));
    CHECK_CUDA(hipFree(dRw));
    CHECK_CUDA(hipFree(dRMv));
    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUDA(hipFree(dBufferR));
    free(epsilon);
    free(static_y);
    return 0;
}

int policy_optimisation(
    int *Pi, // SIZE OF THE INIT POLICY WILL BE P.M
    const int pm,    // TRANSITION COL NUMBER
    const int pn,    // TRANSITION ROW NUMBER 
    const int pnz,   // TRANSITION NON ZERO VALUES
    const int *pi,   // TRANSITION ROW PTR CSR
    const int *pj,   // TRANSITION COL VECTOR CSR
    const float *px, // TRANSITION VALUE VECTOR
    const int rm,    // REWARDS VALUE ROW NUMBER
    const int rn,    // REWARDS VALUE COLS NUMBER
    const int rnz,   // REWARDS NON ZERO VALUES
    const int *ri,   // REWARDS MATRIX ROW PTR CSR
    const int *rj,   // REWARDS MATRIX COL VECTOR CSR
    const float *rx, // REWARDS MATRIX VALUE VECTOR
    float *x,  // Assumes that x is set to the initial value
    float *y,  // TMP ACC VALUE VECTOR
    float *rmv, // initial R vec
    const float *w,  // REPEATED WEIGHT VECTOR
    const float eps,  // THRESHOLD
    int block_size,
    const int *enabled_actions,
    const int *adj_sidx,
    const float *stable,
    int max_iter
){
    /*
    This function is the second part of the value iteration implementation
    */
    hipsparseHandle_t handle = NULL;
    hipsparseCreate(&handle);
    hipblasHandle_t blashandle;
    hipblasCreate(&blashandle);
    hipsparseSpMatDescr_t descrP = NULL;
    hipsparseSpMatDescr_t descrR = NULL;

    // ----------------------------------------------------------------
    //                             POLICY
    // ----------------------------------------------------------------

    int *PI, *EnabledActions, *AdjSIDX;
    float *dStable;
    CHECK_CUDA(hipMalloc((void**)&PI, block_size * sizeof(int)));
    CHECK_CUDA(hipMemcpy(PI, Pi, block_size * sizeof(int), hipMemcpyHostToDevice));
    //
    CHECK_CUDA(hipMalloc((void**)&EnabledActions, block_size * sizeof(int)));
    CHECK_CUDA(hipMemcpy(EnabledActions, enabled_actions, block_size * sizeof(int), hipMemcpyHostToDevice));
    //
    CHECK_CUDA(hipMalloc((void**)&AdjSIDX, block_size * sizeof(int)));
    CHECK_CUDA(hipMemcpy(AdjSIDX, adj_sidx, block_size * sizeof(int), hipMemcpyHostToDevice));
    //
    CHECK_CUDA(hipMalloc((void**)&dStable, block_size * sizeof(float)));
    CHECK_CUDA(hipMemcpy(dStable, stable, block_size * sizeof(float), hipMemcpyHostToDevice));

    // ----------------------------------------------------------------
    //                       Transition Matrix
    // ----------------------------------------------------------------

    //allocate dCsrRowPtr, dCsrColPtr, dCsrValPtr
    int *dPCsrRowPtr, *dPCsrColPtr;
    float *dPCsrValPtr;

    // allocate device memory to store the sparse CSR 
    CHECK_CUDA(hipMalloc((void **)&dPCsrValPtr, sizeof(float) * pnz));
    CHECK_CUDA(hipMalloc((void **)&dPCsrRowPtr, sizeof(int) * (pm + 1)));
    CHECK_CUDA(hipMalloc((void **)&dPCsrColPtr, sizeof(int) * pnz));

    CHECK_CUDA(hipMemcpy(dPCsrValPtr, px, sizeof(float) * pnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dPCsrColPtr, pj, sizeof(int) * pnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dPCsrRowPtr, pi, sizeof(int) * (pm + 1), hipMemcpyHostToDevice));
    
    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrP, // MATRIX DESCRIPTION
        pm, // NUMBER OF ROWS
        pn, // NUMBER OF COLS
        pnz, // NUMBER OF NON ZERO VALUES
        dPCsrRowPtr, // ROWS OFFSETS
        dPCsrColPtr, // COL INDICES
        dPCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));

    // ----------------------------------------------------------------
    //                       Rewards Matrix
    // ----------------------------------------------------------------
    
    int *dRCsrRowPtr, *dRCsrColPtr;
    float *dRCsrValPtr;

    // allocate device memory to store the sparse CSR 
    CHECK_CUDA(hipMalloc((void **)&dRCsrValPtr, sizeof(float) * rnz));
    CHECK_CUDA(hipMalloc((void **)&dRCsrRowPtr, sizeof(int) * (rm + 1)));
    CHECK_CUDA(hipMalloc((void **)&dRCsrColPtr, sizeof(int) * rnz));
    CHECK_CUDA(hipMemcpy(dRCsrValPtr, rx, sizeof(float) * rnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dRCsrColPtr, rj, sizeof(int) * rnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dRCsrRowPtr, ri, sizeof(int) * (rm + 1), hipMemcpyHostToDevice)); 

    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrR, // MATRIX DESCRIPTION
        rm, // NUMBER OF ROWS
        rn, // NUMBER OF COLS
        rnz, // NUMBER OF NON ZERO VALUES
        dRCsrRowPtr, // ROWS OFFSETS
        dRCsrColPtr, // COL INDICES
        dRCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));

    // ----------------------------------------------------------------
    //                      Start of VI
    // ----------------------------------------------------------------

    // --------------TRANSITION MATRIX MULTIPLICATION SETUP------------
    float alpha = 1.0;
    float beta = 1.0;
    float policy_stable = 0.;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecX, vecY;
    //
    float *dX, *dXtmp, *dY, *dStaticY; 
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    CHECK_CUDA(hipMalloc((void**)&dX, block_size * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dY, pm * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dXtmp, block_size * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dStaticY, pm * sizeof(float)));
    //hipMalloc((void**)&d_eps, sizeof(float));

    // create a initial Y vector
    //float *static_y = (float*) calloc(pm, sizeof(float));
    
    // copy the vector from host memory to device memory
    CHECK_CUDA(hipMemcpy(dY, y, pm * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dStaticY, y, pm * sizeof(float), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMemcpy(dX, x, block_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dXtmp, x, block_size * sizeof(float), hipMemcpyHostToDevice));

    // create a dense vector on device memory
    // printf("block size: %i\n", block_size);
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, block_size, dX, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, pm, dY, HIP_R_32F));

    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descrP, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    
    // --------------REWARDS MATRIX MULTIPLICATION SETUP---------------

    float alphaR = 1.0;
    float betaR = 1.0;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecW, vecRMv;
    float *dRw, *dRMv, *dRstaticMx;
    void* dBufferR = NULL;
    size_t bufferSizeR = 0;

    //float *rmv = (float*) calloc(rm, sizeof(float));

    CHECK_CUDA(hipMalloc((void**)&dRw, rn * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dRMv, rm * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dRstaticMx, rm * sizeof(float)));

    // copy the vector from host memory to device memory
    CHECK_CUDA(hipMemcpy(dRw, w, rn * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dRMv, rmv, rm  * sizeof(float), hipMemcpyHostToDevice));
    //hipMemcpy(dRstaticMx, rmv, rm * sizeof(float), hipMemcpyHostToDevice);

    // create a dense vector on device memory
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecW, rn, dRw, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecRMv, rm, dRMv, HIP_R_32F));

    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeR));
    CHECK_CUDA(hipMalloc(&dBufferR, bufferSizeR));

    // ONE OFF REWARDS COMPUTATION

    CHECK_CUSPARSE(hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F, 
        HIPSPARSE_MV_ALG_DEFAULT, dBufferR));
    
    // ALGORITHM LOOP - POLICY GENERATION
    for (int algo_i = 0; algo_i < max_iter; algo_i ++) {

        CHECK_CUSPARSE(hipsparseSpMV(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, descrP, vecX, &beta, vecY, HIP_R_32F, 
            HIPSPARSE_MV_ALG_DEFAULT, dBuffer));    

        // ---------------------SUM DENSE VECTORS-------------------------

        /* 
        i.e. we are summing dY + dRMv
        */
        
        CHECK_CUBLAS(hipblasSaxpy(blashandle, pm, &alpha, dRMv, 1, dY, 1));
        // ------------------COMPUTE POLICY STABLE------------------------
        
        max_value_launcher(dY, EnabledActions, AdjSIDX, dXtmp, dX, PI, dStable, 
                           eps, block_size);
        
        // we can compute if the policy is stable with cublas 
        CHECK_CUBLAS(hipblasScopy(blashandle, block_size, dXtmp, 1, dX, 1));
        
        hipblasSasum(blashandle, block_size, dStable, 1, &policy_stable);
        if (policy_stable == 0) {
            break;
        }
        CHECK_CUBLAS(hipblasScopy(blashandle, pm, dStaticY, 1, dY, 1));
    }

    CHECK_CUDA(hipMemcpy(x, dX, block_size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(Pi, PI, block_size * sizeof(int), hipMemcpyDeviceToHost));
    

    // MEMORY MANAGEMENT
    //destroy the vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(descrP));
    CHECK_CUSPARSE(hipsparseDestroySpMat(descrR));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRMv));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecW));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUBLAS(hipblasDestroy(blashandle));

    // Free the device memory
    CHECK_CUDA(hipFree(dPCsrColPtr));
    CHECK_CUDA(hipFree(dPCsrRowPtr));
    CHECK_CUDA(hipFree(dPCsrValPtr));
    CHECK_CUDA(hipFree(dRCsrColPtr));
    CHECK_CUDA(hipFree(dRCsrRowPtr));
    CHECK_CUDA(hipFree(dRCsrValPtr));
    CHECK_CUDA(hipFree(dX));
    CHECK_CUDA(hipFree(dXtmp));
    CHECK_CUDA(hipFree(dY));
    CHECK_CUDA(hipFree(dStaticY));
    CHECK_CUDA(hipFree(dStable));
    CHECK_CUDA(hipFree(AdjSIDX));
    CHECK_CUDA(hipFree(EnabledActions));
    CHECK_CUDA(hipFree(dRw));
    CHECK_CUDA(hipFree(dRMv));
    CHECK_CUDA(hipFree(dRstaticMx));
    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUDA(hipFree(dBufferR));
    CHECK_CUDA(hipFree(PI));
    //free(epsilon);
    return 0;
}

int policy_value_stream(
    int p_init_m,
    int p_init_n,
    int p_init_nz,
    int *p_init_i,
    int *p_init_j,
    float *p_init_x,
    int p_init_i_size,
    int p_m,
    int p_n,
    int p_nz,
    int *p_i,
    int *p_j,
    float *p_x,
    int p_i_size,
    int r_init_m,
    int r_init_n,
    int r_init_nz,
    int *r_init_i,
    int *r_init_j,
    float *r_init_x,
    int r_init_i_size,
    int r_m,
    int r_n,
    int r_nz,
    int *r_i,
    int *r_j,
    float *r_x,
    int r_i_size,
    float *x_init,
    float *y_init,
    float *w_init,
    float *rmv_init,
    float *y, 
    float *rmv,
    float *w,
    int *unstable,
    int *Pi,
    int *enabled_actions,
    int *adj_sidx,
    float *stable,
    float eps,
    int max_iter,
    int max_unstable
    ) {
    /* 
    Get the COO matrix into sparsescoo fmt

    Then multiply the COO by the initial value vector

    The rewards matrix is also sparse so it will need a sparse matrix descr
    as well. Multiply R by a repeated weight vector in the number 
    of prods and actions

    Finally sum the result

    This should happen in a loop until convergence

    I also want to do some wall timing to see some statistics on 
    the GPU 
    */
    // build the sparse transition matrix first

    hipsparseHandle_t     handle;
    hipblasHandle_t       blashandle;

    //hipEvent_t          start, stop;
    hipStream_t         stream0, stream1;


    hipsparseSpMatDescr_t descrPinit = NULL, descrP = NULL, 
                         descrRinit = NULL, descrR = NULL;

    CHECK_CUSPARSE(hipsparseCreate(&handle));
    CHECK_CUBLAS(hipblasCreate(    &blashandle));
    CHECK_CUDA( hipStreamCreate( &stream0 ) );
    CHECK_CUDA( hipStreamCreate( &stream1 ) );
    // allocated the device memory for the COO matrix
    CHECK_CUDA( hipHostRegister(w_init, r_init_n * sizeof(float), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(w, r_init_n * sizeof(float), hipHostRegisterDefault));

    // declare some pinned memory for the transition matrix
    CHECK_CUDA( hipHostRegister(p_init_i, p_init_i_size * sizeof(int), hipHostRegisterDefault) );
    CHECK_CUDA( hipHostRegister(p_init_j, p_init_nz * sizeof(int), hipHostRegisterDefault) );
    CHECK_CUDA( hipHostRegister(p_init_x, p_init_nz * sizeof(float), hipHostRegisterDefault) );

    CHECK_CUDA( hipHostRegister(p_i, p_i_size * sizeof(int), hipHostRegisterDefault) );
    CHECK_CUDA( hipHostRegister(p_j, p_nz * sizeof(int), hipHostRegisterDefault) );
    CHECK_CUDA( hipHostRegister(p_x, p_nz * sizeof(float), hipHostRegisterDefault) );

    CHECK_CUDA( hipHostRegister(r_init_i, r_init_i_size * sizeof(int), hipHostRegisterDefault) );
    CHECK_CUDA( hipHostRegister(r_init_j, r_init_nz * sizeof(int), hipHostRegisterDefault) );
    CHECK_CUDA( hipHostRegister(r_init_x, r_init_nz * sizeof(float), hipHostRegisterDefault) );
 
    CHECK_CUDA( hipHostRegister(r_i, r_i_size * sizeof(int), hipHostRegisterDefault) );
    CHECK_CUDA( hipHostRegister(r_j, r_nz * sizeof(int), hipHostRegisterDefault) );
    CHECK_CUDA( hipHostRegister(r_x, r_nz * sizeof(float), hipHostRegisterDefault) );

    CHECK_CUDA( hipHostRegister(x_init, p_init_n * sizeof(float), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(y_init, p_init_m * sizeof(float), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(rmv_init, p_init_m * sizeof(float), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(unstable, p_init_m * sizeof(int), hipHostRegisterDefault));

    CHECK_CUDA( hipHostRegister(y, p_m * sizeof(float), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(rmv, p_m * sizeof(float), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(Pi, p_n * sizeof(int), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(enabled_actions, p_n * sizeof(int), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(adj_sidx, p_n * sizeof(int), hipHostRegisterDefault));
    CHECK_CUDA( hipHostRegister(stable, p_n * sizeof(float), hipHostRegisterDefault));
    
    
    // ----------------------------------------------------------------
    //       STREAM 0: DATA TRANSFER
    // ----------------------------------------------------------------

    // ----------------------------------------------------------------
    //                       Initial Policy Transition Matrix
    // ----------------------------------------------------------------
    
    //allocate dCsrRowPtr, dCsrColPtr, dCsrValPtr
    
    int *dPInitCsrRowPtr, *dPInitCsrColPtr;
    float *dPInitCsrValPtr;
    
    // allocate device memory to store the sparse CSR 
    CHECK_CUDA(hipMalloc((void **)&dPInitCsrValPtr, sizeof(float) * p_init_nz));
    CHECK_CUDA(hipMalloc((void **)&dPInitCsrColPtr, sizeof(int) * p_init_nz));
    CHECK_CUDA(hipMalloc((void **)&dPInitCsrRowPtr, sizeof(int) * p_init_i_size));

    // |
    // --------------------> Put all of the init on stream 0
    CHECK_CUDA(hipMemcpyAsync(dPInitCsrValPtr, p_init_x, sizeof(float) * p_init_nz, hipMemcpyHostToDevice, stream0) );
    CHECK_CUDA(hipMemcpyAsync(dPInitCsrColPtr, p_init_j, sizeof(int) * p_init_nz, hipMemcpyHostToDevice, stream0) );
    CHECK_CUDA(hipMemcpyAsync(dPInitCsrRowPtr, p_init_i, sizeof(int) * p_init_i_size, hipMemcpyHostToDevice, stream0) );

    // create the sparse CSR matrix in device memory
    
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrPinit, // MATRIX DESCRIPTION
        p_init_m, // NUMBER OF ROWS
        p_init_n, // NUMBER OF COLS
        p_init_nz, // NUMBER OF NON ZERO VALUES
        dPInitCsrRowPtr, // ROWS OFFSETS
        dPInitCsrColPtr, // COL INDICES
        dPInitCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));

    // ----------------------------------------------------------------
    //                       Initial Rewards Matrix
    // ----------------------------------------------------------------

    int *dRInitCsrRowPtr, *dRinitCsrColPtr;
    float *dRInitCsrValPtr;

    // allocate device memory to store the sparse CSR 
    CHECK_CUDA( hipMalloc((void **)&dRInitCsrValPtr, sizeof(float) * r_init_nz) );
    CHECK_CUDA( hipMalloc((void **)&dRInitCsrRowPtr, sizeof(int) * r_init_i_size) );
    CHECK_CUDA( hipMalloc((void **)&dRinitCsrColPtr, sizeof(int) * r_init_nz) );
    CHECK_CUDA( hipMemcpyAsync(dRInitCsrValPtr, r_init_x, sizeof(float) * r_init_nz, hipMemcpyHostToDevice, stream0) );
    CHECK_CUDA( hipMemcpyAsync(dRinitCsrColPtr, r_init_j, sizeof(int) * r_init_nz, hipMemcpyHostToDevice, stream0) );
    CHECK_CUDA( hipMemcpyAsync(dRInitCsrRowPtr, r_init_i, sizeof(int) * r_init_i_size, hipMemcpyHostToDevice, stream0) );

    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrRinit, // MATRIX DESCRIPTION
        r_init_m, // NUMBER OF ROWS
        r_init_n, // NUMBER OF COLS
        r_init_nz, // NUMBER OF NON ZERO VALUES
        dRInitCsrRowPtr, // ROWS OFFSETS
        dRinitCsrColPtr, // COL INDICES
        dRInitCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));

    // --------------INTIAL TRANSITION MATRIX MULTIPLICATION SETUP------------
    float alpha = 1.0;
    float beta = 1.0;
    float policy_stable = 0.;
    //float *epsilon = (float*) malloc(p_init_m * sizeof(float));
    //int iepsilon;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecXinit, vecYinit;
    float *dXinit, *dYinit, *dZinit, *dStaticYinit;
    int *dUnstableInit;
    void* dBufferInit = NULL;
    size_t bufferSizeInit = 0;

    // Allocate the device memory
    CHECK_CUDA( hipMalloc((void**)&dXinit, p_init_m * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**)&dYinit, p_init_m * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**)&dZinit, p_init_m * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**)&dStaticYinit, p_init_m * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**)&dUnstableInit, p_init_m * sizeof(int)) );

    // Allocate registered memory to the device
    CHECK_CUDA( hipMemcpyAsync(dXinit, x_init, sizeof(float) * p_init_n, hipMemcpyHostToDevice, stream0) );
    CHECK_CUDA( hipMemcpyAsync(dYinit, y_init, sizeof(float) * p_init_m, hipMemcpyHostToDevice, stream0) );
    CHECK_CUDA( hipMemcpyAsync(dUnstableInit, unstable, sizeof(int) * p_init_m, hipMemcpyHostToDevice, stream0) );
    CHECK_CUDA( hipMemcpyAsync(dStaticYinit, y_init, sizeof(float) * p_init_m, hipMemcpyHostToDevice, stream0) );

    // --------------REWARDS MATRIX MULTIPLICATION SETUP---------------
    float alphaR = 1.0;
    float betaR = 1.0;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecWinit, vecRMvinit;
    float *dRwInit, *dRMvInit;
    void* dBufferRinit = NULL;
    size_t bufferSizeRinit = 0;

    //float *rmv = (float*) calloc(rm, sizeof(float));

    CHECK_CUDA( hipMalloc((void**)&dRwInit, r_init_n * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**)&dRMvInit, r_init_m * sizeof(float)) );

    // copy the vector from host memory to device memory
    CHECK_CUDA( hipMemcpyAsync(dRwInit, w_init, r_init_n * sizeof(float), hipMemcpyHostToDevice, stream0) );
    CHECK_CUDA( hipMemcpyAsync(dRMvInit, rmv_init, r_init_m * sizeof(float), hipMemcpyHostToDevice, stream0) );
    
    // ----------------------------------------------------------------
    //       STREAM 0: END OF DATA TRANSFER
    // ----------------------------------------------------------------

    // ----------------------------------------------------------------
    //       STREAM 1: DATA TRANSFER
    // ----------------------------------------------------------------

    // ----------------------------------------------------------------
    //                       Complete Transition Matrix
    // ----------------------------------------------------------------
    
    //allocate dCsrRowPtr, dCsrColPtr, dCsrValPtr
    int *dPCsrRowPtr, *dPCsrColPtr;
    float *dPCsrValPtr;

    // allocate device memory to store the sparse CSR 
    CHECK_CUDA(hipMalloc((void **)&dPCsrValPtr, sizeof(float) * p_nz));
    CHECK_CUDA(hipMalloc((void **)&dPCsrColPtr, sizeof(int) * p_nz));
    CHECK_CUDA(hipMalloc((void **)&dPCsrRowPtr, sizeof(int) * p_i_size));

    // |
    // --------------------> The complete transition matrix => stream1
    CHECK_CUDA(hipMemcpyAsync(dPCsrValPtr, p_x, sizeof(float) * p_nz, hipMemcpyHostToDevice, stream1) );
    CHECK_CUDA(hipMemcpyAsync(dPCsrColPtr, p_j, sizeof(int) * p_nz, hipMemcpyHostToDevice, stream1) );
    CHECK_CUDA(hipMemcpyAsync(dPCsrRowPtr, p_i, sizeof(int) * p_i_size, hipMemcpyHostToDevice, stream1) );

    
    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrP, // MATRIX DESCRIPTION
        p_m, // NUMBER OF ROWS
        p_n, // NUMBER OF COLS
        p_nz, // NUMBER OF NON ZERO VALUES
        dPCsrRowPtr, // ROWS OFFSETS
        dPCsrColPtr, // COL INDICES
        dPCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));
    

    // ----------------------------------------------------------------
    //                       Complete Rewards Matrix
    // ----------------------------------------------------------------

    int *dRCsrRowPtr, *dRCsrColPtr;
    float *dRCsrValPtr;

    // allocate device memory to store the sparse CSR 
    CHECK_CUDA( hipMalloc((void **)&dRCsrValPtr, sizeof(float) * r_nz) );
    CHECK_CUDA( hipMalloc((void **)&dRCsrRowPtr, sizeof(int) * r_i_size) );
    CHECK_CUDA( hipMalloc((void **)&dRCsrColPtr, sizeof(int) * r_nz) );
    CHECK_CUDA( hipMemcpyAsync(dRCsrValPtr, r_x, sizeof(float) * r_nz, hipMemcpyHostToDevice, stream1) );
    CHECK_CUDA( hipMemcpyAsync(dRCsrColPtr, r_j, sizeof(int) * r_nz, hipMemcpyHostToDevice, stream1) );
    CHECK_CUDA( hipMemcpyAsync(dRCsrRowPtr, r_i, sizeof(int) * r_i_size, hipMemcpyHostToDevice, stream1) );

    
    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrR, // MATRIX DESCRIPTION
        r_m, // NUMBER OF ROWS
        r_n, // NUMBER OF COLS
        r_nz, // NUMBER OF NON ZERO VALUES
        dRCsrRowPtr, // ROWS OFFSETS
        dRCsrColPtr, // COL INDICES
        dRCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));
    

    // ----------------------------------------------------------------
    //                             POLICY
    // ----------------------------------------------------------------

    int *PI, *EnabledActions, *AdjSIDX;
    float *dStable;
    CHECK_CUDA(hipMalloc((void**)&PI, p_n * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&EnabledActions, p_n * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&AdjSIDX, p_n * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&dStable, p_n * sizeof(float)));
    //
    CHECK_CUDA(hipMemcpyAsync(PI, Pi, p_n * sizeof(int), hipMemcpyHostToDevice, stream1));
    CHECK_CUDA(hipMemcpyAsync(EnabledActions, enabled_actions, p_n * sizeof(int), hipMemcpyHostToDevice, stream1));
    CHECK_CUDA(hipMemcpyAsync(AdjSIDX, adj_sidx, p_n * sizeof(int), hipMemcpyHostToDevice, stream1));
    CHECK_CUDA(hipMemcpyAsync(dStable, stable, p_n * sizeof(float), hipMemcpyHostToDevice, stream1));
    //
    // --------------TRANSITION MATRIX MULTIPLICATION SETUP------------

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecX, vecY;
    //
    float *dXtmp, *dY, *dStaticY; 
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    CHECK_CUDA(hipMalloc((void**)&dXtmp, p_n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dY, p_m * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dStaticY, p_m * sizeof(float)));
    // copy the vector from host memory to device memory
    CHECK_CUDA(hipMemcpyAsync(dY, y, p_m * sizeof(float), hipMemcpyHostToDevice, stream1));
    CHECK_CUDA(hipMemcpyAsync(dStaticY, y, p_m * sizeof(float), hipMemcpyHostToDevice, stream1));

    // --------------REWARDS MATRIX MULTIPLICATION SETUP---------------

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecW, vecRMv;
    float *dRw, *dRMv;
    void* dBufferR = NULL;
    size_t bufferSizeR = 0;

    //float *rmv = (float*) calloc(rm, sizeof(float));

    CHECK_CUDA(hipMalloc((void**)&dRw, r_n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dRMv, r_m * sizeof(float)));

    // copy the vector from host memory to device memory
    CHECK_CUDA(hipMemcpyAsync(dRw, w, r_n * sizeof(float), hipMemcpyHostToDevice, stream1) );
    CHECK_CUDA(hipMemcpyAsync(dRMv, rmv, r_m  * sizeof(float), hipMemcpyHostToDevice, stream1) );

    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, p_m, dY, HIP_R_32F));

    CHECK_CUSPARSE( hipsparseSetStream(handle, stream1) );

    // create a dense vector on device memory
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecW, r_n, dRw, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecRMv, r_m, dRMv, HIP_R_32F));

    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeR));
    CHECK_CUDA(hipMalloc(&dBufferR, bufferSizeR));

    // ----------------------------------------------------------------
    //       STREAM 1: END OF DATA TRANSFER
    // ----------------------------------------------------------------

    // ----------------------------------------------------------------
    //                      Start of VI
    // ----------------------------------------------------------------
    
    // TRANSITION MATRIX CUSPARSE SETUP

    // create a dense vector on device memory
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecYinit, p_init_m, dYinit, HIP_R_32F) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecXinit, p_init_n, dXinit, HIP_R_32F) );

    // REWARDS MATRIX CUSPARSE SETUP
    //
    //hipMemcpy(dRstaticMx, rmv, rm * sizeof(float), hipMemcpyHostToDevice);

    // create a dense vector on device memory
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecWinit, r_init_n, dRwInit, HIP_R_32F) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecRMvinit, r_init_m, dRMvInit, HIP_R_32F) );
    
    // Allocate the buffers for the matrix-vector multiplication workspace
    //
    // Set the cusparse handle to the stream just before the operation 
    // according to the documentation 
    
    CHECK_CUSPARSE( hipsparseSetStream(handle, stream0) );
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descrPinit, vecXinit, &beta, vecYinit, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeInit) );
    CHECK_CUDA( hipMalloc(&dBufferInit, bufferSizeInit) );
    
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alphaR, descrRinit, vecWinit, &betaR, vecRMvinit, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeRinit) );
    CHECK_CUDA( hipMalloc(&dBufferRinit, bufferSizeRinit) );
    CHECK_CUBLAS( hipblasSetStream(blashandle, stream0) );
    
    
    CHECK_CUSPARSE(hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &alphaR, descrRinit, vecWinit, &betaR, vecRMvinit, HIP_R_32F, 
        HIPSPARSE_MV_ALG_DEFAULT, dBufferRinit));
    
    float maxeps;
    maxeps = 0.0f;

    for (int algo_i = 0; algo_i < max_iter; algo_i ++) {

        CHECK_CUSPARSE(hipsparseSpMV(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, descrPinit, vecXinit, &beta, vecYinit, HIP_R_32F, 
            HIPSPARSE_MV_ALG_DEFAULT, dBufferInit));

        // push this into the algorithm loop


        // ---------------------SUM DENSE VECTORS-------------------------

         
        //The gpu memory should already be allocated, i.e. we are summing
        //dY + dRMv
        
        CHECK_CUBLAS(hipblasSaxpy(blashandle, p_init_m, &alpha, dRMvInit, 1, dYinit, 1));
        
        // ---------------------COMPUTE EPSILON---------------------------
        // what is the difference between dY and dX

        // EPSILON COMPUTATION
        abs_diff_launcher(dYinit, dXinit, dZinit, dUnstableInit, p_init_m, max_unstable);
        //CHECK_CUBLAS(hipblasIsamax(blashandle, pm, dZ, 1, &iepsilon));

        thrust::device_ptr<float> dev_ptr(dZinit);
        maxeps = *thrust::max_element(thrust::device, dev_ptr, dev_ptr + p_init_m);

        CHECK_CUBLAS(hipblasScopy(blashandle, p_init_m, dYinit, 1, dXinit, 1));
        // RESET Y
        CHECK_CUBLAS(hipblasScopy(blashandle, p_init_m, dStaticYinit, 1, dYinit, 1));
        //std::cout << "EPS_TEST " << "THRUST "<< maxeps << std::endl;
        if (maxeps < eps || isnan(maxeps)) {
            //printf("INITIAL POLICY GENERATED; EPS TOL REACHED in %i ITERATIONS\n", algo_i);
            break;
        }
    }

    // --------------STREAM SYNCHRONISATION------------

    CHECK_CUDA( hipDeviceSynchronize() );

    // ----------------POLICY OPTIMISATION-------------

    CHECK_CUDA(hipMemcpy(dXtmp, dXinit, p_n * sizeof(float), hipMemcpyDeviceToDevice));
    CHECK_CUSPARSE( hipsparseSetStream(handle, 0) );

    CHECK_CUSPARSE(hipsparseSpMV(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F, 
        HIPSPARSE_MV_ALG_DEFAULT, dBufferR));

    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descrP, vecXinit, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // ALGORITHM LOOP - POLICY GENERATION
    for (int algo_i = 0; algo_i < max_iter; algo_i ++) {

        CHECK_CUSPARSE(hipsparseSpMV(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alpha, descrP, vecXinit, &beta, vecY, HIP_R_32F, 
            HIPSPARSE_MV_ALG_DEFAULT, dBuffer));    

        // ---------------------SUM DENSE VECTORS-------------------------

        /* 
        i.e. we are summing dY + dRMv
        */
        
        CHECK_CUBLAS(hipblasSaxpy(blashandle, p_m, &alpha, dRMv, 1, dY, 1));
        // ------------------COMPUTE POLICY STABLE------------------------
        
        max_value_launcher(dY, EnabledActions, AdjSIDX, dXtmp, dXinit, PI, dStable, 
                           eps, p_n);
        
        // we can compute if the policy is stable with cublas 
        CHECK_CUBLAS(hipblasScopy(blashandle, p_n, dXtmp, 1, dXinit, 1));
        
        hipblasSasum(blashandle, p_n, dStable, 1, &policy_stable);
        if (policy_stable == 0) {
            break;
        }
        CHECK_CUBLAS(hipblasScopy(blashandle, p_m, dStaticY, 1, dY, 1));
    }

    // COPY THE SOLUTION BACK TO THE HOST
    CHECK_CUDA(hipMemcpy(x_init, dXinit, p_n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(Pi, PI, p_n * sizeof(float), hipMemcpyDeviceToHost));
    
    // Load the the other data on the 
    // 
    // ----------------------------------------------------------------
    //                       Memory Management
    // ----------------------------------------------------------------
    //CHECK_CUDA( hipStreamSynchronize( stream1 ) );
    CHECK_CUDA( hipStreamDestroy( stream0 ));
    CHECK_CUDA( hipStreamDestroy( stream1 ));
    //destroy the vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(descrPinit));
    CHECK_CUSPARSE(hipsparseDestroySpMat(descrP));
    CHECK_CUSPARSE(hipsparseDestroySpMat(descrRinit));
    CHECK_CUSPARSE(hipsparseDestroySpMat(descrR));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRMv));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecW));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecYinit));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRMvinit));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecWinit));
    CHECK_CUSPARSE(hipsparseDestroy(handle));
    CHECK_CUBLAS(hipblasDestroy(blashandle));

    // Free the device memory
    CHECK_CUDA(hipFree(dPInitCsrColPtr));
    CHECK_CUDA(hipFree(dPInitCsrRowPtr));
    CHECK_CUDA(hipFree(dPInitCsrValPtr));
    CHECK_CUDA(hipFree(dRinitCsrColPtr));
    CHECK_CUDA(hipFree(dRInitCsrRowPtr));
    CHECK_CUDA(hipFree(dRInitCsrValPtr));
    CHECK_CUDA(hipFree(dPCsrColPtr));
    CHECK_CUDA(hipFree(dPCsrRowPtr));
    CHECK_CUDA(hipFree(dPCsrValPtr));
    CHECK_CUDA(hipFree(dRCsrColPtr));
    CHECK_CUDA(hipFree(dRCsrRowPtr));
    CHECK_CUDA(hipFree(dRCsrValPtr));

    CHECK_CUDA(hipFree(dXinit));
    CHECK_CUDA(hipFree(dYinit));
    CHECK_CUDA(hipFree(dStaticYinit));
    CHECK_CUDA(hipFree(dZinit));
    CHECK_CUDA(hipFree(dUnstableInit));
    CHECK_CUDA(hipFree(PI));
    CHECK_CUDA(hipFree(EnabledActions));
    CHECK_CUDA(hipFree(AdjSIDX));
    CHECK_CUDA(hipFree(dStable));
    CHECK_CUDA(hipFree(dXtmp));
    CHECK_CUDA(hipFree(dY));
    CHECK_CUDA(hipFree(dStaticY));
    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUDA(hipFree(dRw));
    CHECK_CUDA(hipFree(dRwInit));
    CHECK_CUDA(hipFree(dRMv));
    CHECK_CUDA(hipFree(dBufferR));

    CHECK_CUDA( hipHostUnregister(p_init_i) );
    CHECK_CUDA( hipHostUnregister(p_init_j) );
    CHECK_CUDA( hipHostUnregister(p_init_x) );
    CHECK_CUDA( hipHostUnregister(p_i) );
    CHECK_CUDA( hipHostUnregister(p_j) );
    CHECK_CUDA( hipHostUnregister(p_x) );

    CHECK_CUDA( hipHostUnregister(r_init_i) );
    CHECK_CUDA( hipHostUnregister(r_init_j) );
    CHECK_CUDA( hipHostUnregister(r_init_x) );
    CHECK_CUDA( hipHostUnregister(r_i) );
    CHECK_CUDA( hipHostUnregister(r_j) );
    CHECK_CUDA( hipHostUnregister(r_x) );

    CHECK_CUDA( hipHostUnregister(x_init) );
    CHECK_CUDA( hipHostUnregister(w_init) );
    CHECK_CUDA( hipHostUnregister(w) );
    CHECK_CUDA( hipHostUnregister(y_init) );
    CHECK_CUDA( hipHostUnregister(rmv_init) );
    CHECK_CUDA( hipHostUnregister(unstable) );
    CHECK_CUDA( hipHostUnregister(y) );
    CHECK_CUDA( hipHostUnregister(rmv) );
    CHECK_CUDA( hipHostUnregister(Pi) );
    CHECK_CUDA( hipHostUnregister(enabled_actions ) );
    CHECK_CUDA( hipHostUnregister(adj_sidx) );
    CHECK_CUDA( hipHostUnregister(stable) );
    //hipFree(d_eps);
    return 0;
}


int multi_obj_solution(
    int pm,
    int pn,
    int pnz,
    int * pi,
    int * pj,
    float * px,
    int pi_size,
    int rm,
    int rn,
    int rnz,
    int *ri,
    int *rj,
    float *rx,
    int ri_size,
    float eps,
    int nobjs,
    float *x,
    float *w,
    float *z,
    int *unstable,
    int max_iter, 
    int max_unstable
) {
    // Setup the framework infrastructure
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    hipblasHandle_t blashandle;
    CHECK_CUBLAS(hipblasCreate(&blashandle));


    hipsparseSpMatDescr_t descrP = NULL;
    hipsparseSpMatDescr_t descrR = NULL;

    //float rStorage[rm * nobjs] = { 0. };

    // allocated the device memory for the COO matrix

    // ----------------------------------------------------------------
    //                       Transition Matrix
    // ----------------------------------------------------------------

    //allocate dCsrRowPtr, dCsrColPtr, dCsrValPtr
    int *dPCsrRowPtr, *dPCsrColPtr;
    float *dPCsrValPtr;

    // allocate device memory to store the sparse CSR 
    hipMalloc((void **)&dPCsrValPtr, sizeof(float) * pnz);
    hipMalloc((void **)&dPCsrRowPtr, sizeof(int) * pi_size);
    hipMalloc((void **)&dPCsrColPtr, sizeof(int) * pnz);

    hipMemcpy(dPCsrValPtr, px, sizeof(float) * pnz, hipMemcpyHostToDevice);
    hipMemcpy(dPCsrColPtr, pj, sizeof(int) * pnz, hipMemcpyHostToDevice);
    hipMemcpy(dPCsrRowPtr, pi, sizeof(int) * pi_size, hipMemcpyHostToDevice);
    
    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrP, // MATRIX DESCRIPTION
        pm, // NUMBER OF ROWS
        pn, // NUMBER OF COLS
        pnz, // NUMBER OF NON ZERO VALUES
        dPCsrRowPtr, // ROWS OFFSETS
        dPCsrColPtr, // COL INDICES
        dPCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));
    
    // ----------------------------------------------------------------
    //                       Rewards Matrix
    // ----------------------------------------------------------------
    
    int *dRCsrRowPtr, *dRCsrColPtr;
    float *dRCsrValPtr;

    // allocate device memory to store the sparse CSR 
    hipMalloc((void **)&dRCsrValPtr, sizeof(float) * rnz);
    hipMalloc((void **)&dRCsrRowPtr, sizeof(int) * ri_size);
    hipMalloc((void **)&dRCsrColPtr, sizeof(int) * rnz);
    hipMemcpy(dRCsrValPtr, rx, sizeof(float) * rnz, hipMemcpyHostToDevice);
    hipMemcpy(dRCsrColPtr, rj, sizeof(int) * rnz, hipMemcpyHostToDevice);
    hipMemcpy(dRCsrRowPtr, ri, sizeof(int) * ri_size, hipMemcpyHostToDevice);

    // create the sparse CSR matrix in device memory
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &descrR, // MATRIX DESCRIPTION
        rm, // NUMBER OF ROWS
        rn, // NUMBER OF COLS
        rnz, // NUMBER OF NON ZERO VALUES
        dRCsrRowPtr, // ROWS OFFSETS
        dRCsrColPtr, // COL INDICES
        dRCsrValPtr, // VALUES
        HIPSPARSE_INDEX_32I, // INDEX TYPE ROWS
        HIPSPARSE_INDEX_32I, // INDEX TYPE COLS
        HIPSPARSE_INDEX_BASE_ZERO, // BASE INDEX TYPE
        HIP_R_32F // DATA TYPE
    ));

    // ----------------------------------------------------------------
    //                      Start of VI
    // ----------------------------------------------------------------

    // --------------TRANSITION MATRIX MULTIPLICATION SETUP------------
    
    float alpha = 1.0;
    float beta = 1.0;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecX, vecY;
    float *dX, *dY, *dZ, *dStorage;
    int *dUnstable;
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    // Allocate the device memory
    CHECK_CUDA( hipMalloc((void**)&dX, pm * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**)&dY, pm * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**)&dZ, pm * nobjs * sizeof(float)) ); // use this to store the epsilon values
    CHECK_CUDA( hipMalloc((void**)&dStorage, pm * nobjs * sizeof(float)) );
    CHECK_CUDA( hipMalloc((void**)&dUnstable, pm * nobjs * sizeof(int)) );
    
    // copy the vector from host memory to device memory
    CHECK_CUDA(hipMemcpy(dX, x, pn * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dY, dX, pm * sizeof(float), hipMemcpyDeviceToDevice) );
    CHECK_CUDA(hipMemcpy(dZ, z, pm * nobjs * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA(hipMemcpy(dStorage, dZ, nobjs * pm * sizeof(float), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(dUnstable, unstable, nobjs * pm * sizeof(float), hipMemcpyHostToDevice));

    // create a dense vector on device memory
    hipsparseCreateDnVec(&vecX, pn, dX, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, pm, dY, HIP_R_32F);

    hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, descrP, vecX, &beta, vecY, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
    
    // --------------REWARDS MATRIX MULTIPLICATION SETUP---------------

    float alphaR = 1.0;
    float betaR = 1.0;

    // assign the cuda memory for the vectors
    hipsparseDnVecDescr_t vecW, vecRMv;
    float *dRw, *dRMv, *dRStorage;
    void* dBufferR = NULL;
    size_t bufferSizeR = 0;

    //float *rmv = (float*) calloc(rm, sizeof(float));

    CHECK_CUDA( hipMalloc((void**)&dRw, rn * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**)&dRMv, rm * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**)&dRStorage, rm * nobjs * sizeof(float)) )

    // copy the vector from host memory to device memory
    CHECK_CUDA( hipMemcpy(dRw, w, rn * sizeof(float), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipMemcpy(dRMv, dX, rm * sizeof(float), hipMemcpyDeviceToDevice) );
    CHECK_CUDA( hipMemcpy(dRStorage, dZ, rm * nobjs * sizeof(float), hipMemcpyDeviceToDevice) );
    // create a dense vector on device memory
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecW, rn, dRw, HIP_R_32F) );
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecRMv, rm, dRMv, HIP_R_32F) );

    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F,
        HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeR) );
    CHECK_CUDA( hipMalloc(&dBufferR, bufferSizeR) );

    for (int k = 0; k < nobjs; k++) {
        // reset the W vector to all zeros
        CHECK_CUDA( hipMemset(dRw, 0., nobjs * sizeof(float)) )
        CHECK_CUDA( hipMemset(dRMv, 0., rm * sizeof(float)) )
        // Change the value of the w array according to the objective we are
        // considering
        change_elem<<<1, 1>>>(dRw, k, 1.0);
        CHECK_CUSPARSE(hipsparseSpMV(
            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
            &alphaR, descrR, vecW, &betaR, vecRMv, HIP_R_32F, 
            HIPSPARSE_MV_ALG_DEFAULT, dBufferR));
        // copy the relevant values to the Rstorage array in the range of |S|
        copy_elems_launcher(dRStorage, k * rm, dRMv, 0,  rm);
    }

    float maxeps;
    maxeps = 0.0f;

    for (int i = 0; i < max_iter; i++) {

        for (int k = 0; k < nobjs; k++) {
            copy_elems_launcher(dY, 0, dRStorage, k * rm, rm);
            copy_elems_launcher(dX, 0, dStorage, k * pm, pm);
            // The next line compute R(k) + P.x
            /*
            CHECK_CUDA(hipMemcpy(x, dX, pm * sizeof(float), hipMemcpyDeviceToHost));
            printf("k=%i\n", k);
            for (int i=0; i<pm; i++) {
                printf("%f, ", x[i]);
            }
            printf("\n");
            */
            CHECK_CUSPARSE(hipsparseSpMV(
                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                &alpha, descrP, vecX, &betaR, vecY, HIP_R_32F, 
                HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

            // update the epsilons for the computed values
            CHECK_CUDA(hipMemcpy(x, dY, pm * sizeof(float), hipMemcpyDeviceToHost));
            /*
            printf("Y; k=%i\n", k);
            for (int i=0; i<pm; i++) {
                printf("%f, ", x[i]);
            }
            printf("\n");
            */
            mobj_abs_diff_launcher(dX, dY, dZ, dUnstable, k, pm, max_unstable);

            // copy x <- y
            copy_elems_launcher(dStorage, k * pm, dY, 0, pm);
        }
        
        //CHECK_CUDA(hipMemcpy(unstable, dUnstable, pm * nobjs * sizeof(int), hipMemcpyDeviceToHost));
        //printf("\n");
        //for (int i=0; i<pm *nobjs; i++) {
        //    printf("%i, ", unstable[i]);
        //}
        //printf("\n");
        
        // lets try and see if we can access our Z values
        thrust::device_ptr<float> dev_ptr(dZ);
        maxeps = *thrust::max_element(thrust::device, dev_ptr, dev_ptr + pm * nobjs);
        //std::cout << "EPS_TEST " << "THRUST "<< maxeps << std::endl;
        if (maxeps < eps || isnan(maxeps) || isinf(maxeps)) {
            //printf("\nFinished M_obj VPI in %i steps\n", i);
            break;
        }
    }

    CHECK_CUDA(hipMemcpy(z, dStorage, pm * nobjs * sizeof(float), 
                          hipMemcpyDeviceToHost));
    CHECK_CUSPARSE( hipsparseDestroySpMat(descrP) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(descrR) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecRMv) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecW) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    CHECK_CUBLAS( hipblasDestroy(blashandle) )

    // Free the device memory
    CHECK_CUDA( hipFree(dPCsrColPtr) )
    CHECK_CUDA( hipFree(dPCsrRowPtr) )
    CHECK_CUDA( hipFree(dPCsrValPtr) )
    CHECK_CUDA( hipFree(dRCsrColPtr) )
    CHECK_CUDA( hipFree(dRCsrRowPtr) )
    CHECK_CUDA( hipFree(dRCsrValPtr) )
    //hipFree(d_eps);
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    CHECK_CUDA( hipFree(dRStorage) )
    CHECK_CUDA( hipFree(dUnstable) )
    CHECK_CUDA( hipFree(dZ) )
    CHECK_CUDA( hipFree(dStorage) )
    CHECK_CUDA( hipFree(dRw) )
    CHECK_CUDA( hipFree(dRMv) )
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dBufferR) )
    //free(x); free(y); free(rmv); free(w); free(unstable);
    return 0;
}

}